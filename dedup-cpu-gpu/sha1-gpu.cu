#include "hip/hip_runtime.h"
#include<cstdio>
//#include<time.h>
#include<string.h>
#include<unistd.h>
#include<stdlib.h>
#include "gpu-process.h"

#define FROM_BIG_ENDIAN(v)                                          \
 ((v & 0xff) << 24) | ((v & 0xff00) << 8) | ((v & 0xff0000) >> 8) |  \
		((v & 0xff000000) >> 24)                              \

#define LEFTROL( v, n)  (v << n) | (v >> (32 - n))
//#define FINGERPRINT_LEN 20
#define MAX_DATA_LEN 8192

unsigned char *gpu_input_data_s,*gpu_output_data_s;
unsigned int *gpu_offset;

__device__ void GPU_sha1_kernel( unsigned char* data_tmp, unsigned int length_tmp,  unsigned int* md)
{

	unsigned int words[80];
	unsigned int H0 = 0x67452301,	H1 = 0xEFCDAB89, H2 = 0x98BADCFE, H3 = 0x10325476, H4 = 0xC3D2E1F0;
	unsigned int a, b, c, d, e, f, k, temp;
	unsigned int i, j;

	unsigned char add_data[8512];
	int kk;
	long long tmp;
	for(kk=0;kk<length_tmp;kk++)
		add_data[kk]=data_tmp[kk];
	if(length_tmp%64<56)
	{
		add_data[kk++]=0x80;
		int t=length_tmp%64+1;
		for(;t<56;t++)
		{
			add_data[kk++]=0x00;
		}
		tmp=length_tmp-(length_tmp%64)+64;
	}else if(length_tmp%64>56)
	{
		add_data[kk++]=0x80;
		int t=length_tmp%64+1;
		for(;t<64;t++)
		{
			add_data[kk++]=0x00;
		}
		for(t=0;t<56;t++)
		{
			add_data[kk++]=0x00;
		}
		tmp=length_tmp-(length_tmp%64)+128;
	}
	add_data[tmp-8]=(tmp & 0xFF00000000000000) >> 56;
	add_data[tmp-7]=(tmp & 0x00FF000000000000) >> 48;
	add_data[tmp-6]=(tmp & 0x0000FF0000000000) >> 40;
	add_data[tmp-5]=(tmp & 0x000000FF00000000) >> 32;
	add_data[tmp-4]=(tmp & 0x00000000FF000000) >> 24;
	add_data[tmp-3]=(tmp & 0x0000000000FF0000) >> 16;
	add_data[tmp-2]=(tmp & 0x000000000000FF00) >> 8;
	add_data[tmp-1]=(tmp & 0x00000000000000FF);

	unsigned int *data=(unsigned int*)add_data;
	unsigned int dataLen=tmp;

	for(j = 0; j < dataLen; j += 64)
	{
		a = H0;
		b = H1;
		c = H2;
		d = H3;
		e = H4;

		for (i=0; i<16; i++)
		{  
			temp = *(( unsigned int*)(data + j/4+i));
			words[i] = FROM_BIG_ENDIAN(temp);
			
			f = (b & c) | ((~b) & d); 
			k = 0x5A827999;
			temp = LEFTROL(a, 5);
			temp += f + e + k + words[i];
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		
		for (i=16; i<20; i++)
		{
			words[i] = LEFTROL( (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]), 1);
			f = (b & c) | ((~b) & d);
			temp = LEFTROL(a, 5);
			temp += f + e + k  + words[i];
		    	e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		for (i=20; i<40; i++)
		{
  		  words[i] = LEFTROL( (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]), 1);
			f=b ^ c ^ d;
			k= 0x6ED9EBA1;
			temp = LEFTROL(a, 5);
			temp += f + e + k + words[i];
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp; 
		}

		for (i=40; i<60; i++)
		{
		    words[i] = LEFTROL( (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]), 1);
			f = (b & c) | (b & d) | (c & d);
			k = 0x8F1BBCDC;
			temp = LEFTROL(a, 5);
			temp += f + e + k+ words[i];
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		for (i=60; i<80; i++)
		{
		    words[i] = LEFTROL( (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]), 1);
			f = b ^ c ^ d;
			k = 0xCA62C1D6;
			temp = LEFTROL(a, 5);
			temp += f + e + k + words[i];
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		H0 += a;
		H1 += b;
		H2 += c;
		H3 += d;
		H4 += e;
	}

		a = H0;
		b = H1;
		c = H2;
		d = H3;
		e = H4;

	words[0] = FROM_BIG_ENDIAN(128);
	f = (b & c) | ((~b) & d);
	k = 0x5A827999;
	temp = LEFTROL(a, 5);
	temp += f + e + k + words[0];
	e = d;
	d = c;
	c = LEFTROL(b, 30);
	b = a;
	a = temp;

	for (i=1; i<15; i++)
	{   
		words[i] = 0;
		f = (b & c) | ((~b) & d);
		temp = LEFTROL(a, 5);
		temp += f + e + k + words[i];
		e = d;
		d = c;
		c = LEFTROL(b, 30);
		b = a;
		a = temp;
	}	
	

	words[15] =  dataLen*8; 
	f = (b & c) | ((~b) & d);
	temp = LEFTROL(a, 5);
	temp += f + e + k + words[15];
	e = d;
	d = c;
	c = LEFTROL(b, 30);
	b = a;
	a = temp;
	
		for (i=16; i<20; i++)
		{
			words[i] = LEFTROL( (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]), 1);
			f = (b & c) | ((~b) & d);
			temp = LEFTROL(a, 5);
			temp += f + e + k + words[i];
		    	e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		for (i=20; i<40; i++)
		{
		    words[i] = LEFTROL( (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]), 1);
			f=b ^ c ^ d;
			k = 0x6ED9EBA1;
			temp = LEFTROL(a, 5);
			temp += f + e + k + words[i];
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp; 
		}

		for (i=40; i<60; i++)
		{
			words[i] = LEFTROL( (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]), 1);
			f = (b & c) | (b & d) | (c & d);
			k = 0x8F1BBCDC;
			temp = LEFTROL(a, 5);
			temp += f + e + k + words[i];
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		for (i=60; i<80; i++)
		{
			words[i] = LEFTROL( (words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16]), 1);
			f = b ^ c ^ d;
			k = 0xCA62C1D6;
			temp = LEFTROL(a, 5);
			temp += f + e + k + words[i];
			e = d;
			d = c;
			c = LEFTROL(b, 30);
			b = a;
			a = temp;
		}

		H0 += a;
		H1 += b;
		H2 += c;
		H3 += d;
		H4 += e;

		 
		int ct=0;
	md[ct++] =FROM_BIG_ENDIAN( H0);
	md[ct++] =FROM_BIG_ENDIAN( H1);
	md[ct++] =FROM_BIG_ENDIAN( H2);
	md[ct++] =FROM_BIG_ENDIAN( H3);
	md[ct++] =FROM_BIG_ENDIAN( H4);
 
}

__global__ void sha1_kernel(unsigned int *offset, unsigned char *input, unsigned char *output, unsigned int num) 
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<num)
	{
		GPU_sha1_kernel(input+offset[index],offset[index+1]-offset[index],(unsigned int*)(output+index*FINGERPRINT_LEN));
	}
}

//the length of the data, the number of the blocks
void GPU_sha1_init(unsigned int len,unsigned int num)
{
	hipSetDevice(0);
	hipMalloc((void**)&gpu_input_data_s, len*sizeof(unsigned char));
	hipMalloc((void**)&gpu_output_data_s, num*FINGERPRINT_LEN);
	hipMalloc((void**)&gpu_offset, (num+1)*sizeof(unsigned int));
}

void GPU_sha1_destroy(void)
{
	hipFree(gpu_input_data_s);
	hipFree(gpu_output_data_s);
	hipFree(gpu_offset);
}

// the max length of the block is no more than 8192(8K)
void GPU_sha1(unsigned char *input,unsigned char *output,unsigned int *offset,unsigned int num,unsigned int len)
{
	GPU_sha1_init(len,num);
	hipMemcpy(gpu_input_data_s,input,len*sizeof(unsigned char),hipMemcpyHostToDevice);
	hipMemcpy(gpu_offset,offset,(num+1)*sizeof(unsigned int),hipMemcpyHostToDevice);

	unsigned int threadNum=32;
	unsigned int blockNum=(unsigned int)(num+threadNum-1)/threadNum;
	dim3 grid(blockNum,1,1);
	dim3 threads(threadNum,1,1);
	sha1_kernel<<<grid,threads>>>(gpu_offset,gpu_input_data_s,gpu_output_data_s,num);
	hipDeviceSynchronize();

	hipMemcpy(output,gpu_output_data_s,num*FINGERPRINT_LEN,hipMemcpyDeviceToHost);
	GPU_sha1_destroy();
//	hipDeviceSynchronize();
}



/*
void create_file(unsigned int len,unsigned int &num,unsigned int *chunk_offset,unsigned char *str)
{
	srand(time(NULL));	
	for(int i=0;i<len;i++)
		str[i]='a'+rand()%26;
	int next_off,offset=0;
	num=0;	
	while(offset<len)
	{
		chunk_offset[num++]=offset;
		next_off=rand()%8192;
		offset+=next_off;
	}
	chunk_offset[num]=len;
}

int main(int argc,char** argv)
{
	unsigned int len,num;
	unsigned char *input_str,*output;
	unsigned int *chunk_offset=new unsigned int[10000];
	int ch=getopt(argc,argv,"cr:");
	if(ch=='c')
	{
		printf("input len:\n");
		scanf("%d",&len);
		input_str=new unsigned char[len];
		create_file(len,num,chunk_offset,input_str);
		printf("total chunks: %d\n",num);
		output=new unsigned char[(num+1)*FINGERPRINT_LEN];
		for(int i=0;i<num;i++)
		{
			printf("%d  ",chunk_offset[i]);
			if((i+1)%10==0) printf("\n");
		}
		printf("\n");
	}else if(ch=='r')
	{
		//printf("%s\n",optarg);
		FILE *in=fopen(optarg,"r");
		fscanf(in,"%d",&len);
		printf("%d\n",len);
		input_str=new unsigned char[len];
		char tmp=fgetc(in);
		for(int i=0;i<len;i++)
		{
			fscanf(in,"%c",&input_str[i]);
			//printf(":%c",input_str[i]);
		}
		printf("%s\n",input_str);
		fscanf(in,"%d",&num);
		printf("%d\n",num);
		output=new unsigned char[(num+1)*FINGERPRINT_LEN];
		for(int i=0;i<num;i++)
		{
			fscanf(in,"%d",&chunk_offset[i]);
		}
		chunk_offset[num]=len;
		for(int i=0;i<num;i++)
		{
			printf("%d ",chunk_offset[i]);
			if((i+1)%10==0) printf("\n");
		}
		printf("\n");
	}
	//FILE *result=fopen("result.txt","w");
	GPU_sha1_init(len,num);
	GPU_sha1(input_str,output,chunk_offset,num,len);
	unsigned int *sha1_int=(unsigned int*)output;
	int j=0;
	for(int i=0;i<=num;i++)
	{
		printf("%d: ",i);
		for(int k=0;k<5;k++)
			printf("%08x ",sha1_int[j++]);
		printf("\n");
	}
	free(input_str);
	free(output);
	free(chunk_offset);
	GPU_sha1_destroy();
	return 0;
}
*/
